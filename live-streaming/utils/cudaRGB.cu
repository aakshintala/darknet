#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaRGB.h"

//-------------------------------------------------------------------------------------------------------------------------

__global__ void RGBToRGBAf(uchar3* srcImage,
                           float4* dstImage,
                           int width, int height)
{
	const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	const int pixel = y * width + x;

	if( x >= width )
		return;

	if( y >= height )
		return;

//	printf("cuda thread %i %i  %i %i pixel %i \n", x, y, width, height, pixel);

	const float  s  = 1.0f;
	const uchar3 px = srcImage[pixel];

	dstImage[pixel] = make_float4(px.x * s, px.y * s, px.z * s, 255.0f * s);
}

hipError_t cudaRGBToRGBAf( uchar3* srcDev, float4* destDev, size_t width, size_t height )
{
	if( !srcDev || !destDev )
		return hipErrorInvalidDevicePointer;

	const dim3 blockDim(8,8,1);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y), 1);

	RGBToRGBAf<<<gridDim, blockDim>>>( srcDev, destDev, width, height );

	return CUDA(hipGetLastError());
}

//-------------------------------------------------------------------------------------------------------------------------

__global__ void RGBToBGRA8(float3* srcImage,
							uchar4* dstImage,
							int width, int height,
							float scaling_factor)
{
	const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	const int pixel = y * width + x;

	if( x >= width )
		return;

	if( y >= height )
		return;

	const float3 px = srcImage[pixel];
	dstImage[pixel] = make_uchar4(px.z * scaling_factor,
							px.y * scaling_factor,
							px.x * scaling_factor,
							255.0f * scaling_factor);
}

hipError_t cudaRGBToBGRA8( float3* srcDev, uchar4* destDev, size_t width, size_t height, const float2& inputRange, hipStream_t *stream )
{
	if( !srcDev || !destDev )
		return hipErrorInvalidDevicePointer;

	if( width == 0 || height == 0 )
		return hipErrorInvalidValue;

	const float multiplier = 255.0f / inputRange.y;

	const dim3 blockDim(8,8,1);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y), 1);

	RGBToBGRA8<<<gridDim, blockDim, 0, *stream>>>( srcDev, destDev, width, height, multiplier );

	return CUDA(hipGetLastError());
}

hipError_t cudaRGBToBGRA8( float3* srcDev, uchar4* destDev, size_t width, size_t height, hipStream_t *stream)
{
	return cudaRGBToBGRA8(srcDev, destDev, width, height, make_float2(0.0f, 255.0f), *stream);
}


